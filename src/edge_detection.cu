
#include <hip/hip_runtime.h>
#ifdef CUDA
/** 
 * @file edge_detection.cu
 * Routines for VR edge detection
 */
#include "cuda_runtime.h"
#include "device_launch_parameters.h"
#include "cuda_profiler_api.h"
#include "math_constants.h"

#include <stdio.h>
#include "image.h"
#include "math.h"

 /** 
 * @brief Implements the VR edge detector. Eq.2 from :
DOI: 10.1109/83.217230
SourcePubMed
Authors:
Panos E. Trahanias
32.18Foundation for Research and Technology - Hellas
A N Venetsanopoulos
 * @param[in] in_img Image pointer { rgb }
 * @param[in] treshold Cut off for edge or not
 *
 * @return Pointer to the binary edge image or NULL
 *
 * @author Kusnik Damian 
 * @date 26.10.2020
 */
__global__
void CUDA_detect_edge_VR(int* in_data, float* out_data, int width, int height, int threshold) {
	int ic = blockIdx.y * blockDim.y + threadIdx.y;
	int ir = blockIdx.x * blockDim.x + threadIdx.x;
	if (ic >= width - 1 || ir >= height - 1 || ic < 1 || ir < 1)
		return;

	int pos = ir * width + ic;

	int f = 1;
	int iwr, iwc;
	int win_count = 9;
	int center_pix = win_count / 2;
	int offset;
	int min_dist;
	int max_dist;
	int min_dist_index, max_dist_index;
	float dist_sum = 0;

	// this should be dynamic, speed reasons
	int red[9];
	int blue[9];
	int green[9];
	int count = 0;

	float dist_mat[9][9];
	
	for (iwr = 0; iwr < win_count; iwr++)
		for (iwc = 0; iwc < win_count; iwc++)
			dist_mat[9][9] = 0;
	
	/* For each window row */
	for (iwr = -f; iwr <= f; iwr++)
	{
		/* For each window column */
		for (iwc = -f; iwc <= f; iwc++)
		{
			/* Store the red, green, blue values */
			offset = pos +iwr * width + iwc;
			red[count] = (in_data[offset] & 0XFF0000) >> 16;
			green[count] = (in_data[offset] & 0XFF00) >> 8;
			blue[count] = (in_data[offset] & 0XFF) ;
			count++;
		}
	}

	/* Calculate the distances between pairwise color vectors */
	for (iwr = 0; iwr < win_count; iwr++)
	{
		for (iwc = iwr + 1; iwc < win_count; iwc++)
		{
			dist_mat[iwr][iwc] = sqrtf(
				(red[iwr] - red[iwc]) * (red[iwr] - red[iwc]) +
				(green[iwr] - green[iwc]) * (green[iwr] - green[iwc]) +
				(blue[iwr] - blue[iwc]) * (blue[iwr] - blue[iwc])
			);
		}
	}

	/* Calculate the cumulative distance for each pixel and find the minimum */
	min_dist = INT_MAX;
	min_dist_index = center_pix;
	max_dist = INT_MIN;
	max_dist_index = center_pix;
	for (iwr = 0; iwr < win_count; iwr++)
	{
		dist_sum = 0.0;

		for (iwc = 0; iwc < iwr; iwc++)
		{
			dist_sum += dist_mat[iwc][iwr];
		}

		for (iwc = iwr + 1; iwc < win_count; iwc++)
		{
			dist_sum += dist_mat[iwr][iwc];
		}

		if (dist_sum < min_dist)
		{
			min_dist = dist_sum;
			min_dist_index = iwr;
		}
		if (dist_sum > max_dist)
		{
			max_dist = dist_sum;
			max_dist_index = iwr;
		}
	}
	
	out_data[pos] = sqrtf(
			(red[min_dist_index] - red[max_dist_index]) * (red[min_dist_index] - red[max_dist_index]) +
			(green[min_dist_index] - green[max_dist_index]) * (green[min_dist_index] - green[max_dist_index]) +
			(blue[min_dist_index] - blue[max_dist_index]) * (blue[min_dist_index] - blue[max_dist_index])
		);
	return;
}


Image*
detect_edge_VR ( const Image * in_img, const int threshold)
{
 SET_FUNC_NAME ( "detect_edge_VR" );

 byte*** in_data;
 byte** out_data;
 int num_rows, num_cols;
 Image* out_img;
 if ( !is_rgb_img ( in_img ) )
  {
   ERROR_RET ( "Not a color image !", NULL );
  }

 if ( !IS_POS ( threshold ) )
  {
   ERROR ( "theshold ( %d ) must be positive !", threshold );
   return NULL;
  }

 num_rows = get_num_rows(in_img);
 num_cols = get_num_cols(in_img);

 in_data = (byte***)get_img_data_nd(in_img);
 out_img = alloc_img(PIX_GRAY, num_rows, num_cols);
 out_data = (byte**)get_img_data_nd(out_img);

 size_t size_i = size_t(num_rows * num_cols) * sizeof(int);
 size_t size_f = size_t(num_rows * num_cols) * sizeof(float);

 int* int_in_data = (int*)malloc(size_i);

 for (int i = 0; i < num_rows; i++) 
	 for (int j = 0; j < num_cols; j++)
		 int_in_data[i * num_cols + j] = (((int)in_data[i][j][0]) << 16) | ((int)in_data[i][j][1] << 8) | ((int)in_data[i][j][2]);

 int* d_in_data;
 cudaMalloc((void**)&d_in_data, size_i);
 cudaMemcpy(d_in_data, int_in_data, size_i, cudaMemcpyHostToDevice);

 float* d_out_data;
 cudaMalloc((void**)&d_out_data, size_f);

 dim3 blockDim(1, 128, 1);
 dim3 gridDim((unsigned int)ceil((float)num_rows / (float)blockDim.x),
	 (unsigned int)ceil((float)num_cols / (float)blockDim.y),
	 1);

 CUDA_detect_edge_VR << < gridDim, blockDim >> > (d_in_data, d_out_data, num_cols, num_rows, threshold);

 cudaDeviceSynchronize();

 float* float_out_data = (float*)malloc(size_f);
 cudaMemcpy(float_out_data, d_out_data, size_f, cudaMemcpyDeviceToHost);

 for (int i = 0; i < num_rows; i++)
	 for (int j = 0; j < num_cols; j++)
		 out_data[i][j] = (int)(float_out_data[i * num_cols + j])&0xFF;

 // Free device memory

 cudaFree(d_in_data);
 cudaFree(d_out_data);
 cudaDeviceSynchronize();

 free(int_in_data);
 free(float_out_data);

 return out_img;
}
#endif
